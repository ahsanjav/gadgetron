#include "hip/hip_runtime.h"
#include "GadgetronCuException.h"
#include "complext.h"
#include "cuNDArray_blas.h"
#include "cudaDeviceManager.h"

#include <hipblas.h>

namespace Gadgetron {

#define CUBLAS_CALL(fun)                                                                                               \
    {                                                                                                                  \
        hipblasStatus_t err = fun;                                                                                      \
        if (err != HIPBLAS_STATUS_SUCCESS) {                                                                            \
            throw cuda_error(gadgetron_getCublasErrorString(err));                                                     \
        }                                                                                                              \
    }

template <class T>
hipblasStatus_t cublas_axpy(hipblasHandle_t hndl, int n, const T* a, const T* x, int incx, T* y, int incy);
template <class T> hipblasStatus_t cublas_dot(hipblasHandle_t, int, const T*, int, const T*, int, T*, bool cc = true);
template <class T> hipblasStatus_t cublas_nrm2(hipblasHandle_t, int, const T*, int, typename realType<T>::Type* result);
template <class T> hipblasStatus_t cublas_amax(hipblasHandle_t handle, int n, const T* x, int incx, int* result);
template <class T> hipblasStatus_t cublas_amin(hipblasHandle_t handle, int n, const T* x, int incx, int* result);
template <class T>
hipblasStatus_t cublas_asum(hipblasHandle_t handle, int n, const T* x, int incx, typename realType<T>::Type* result);

// NRM2
template <> hipblasStatus_t cublas_nrm2<float>(hipblasHandle_t hndl, int n, const float* x, int inc, float* res) {
    return hipblasSnrm2(hndl, n, x, inc, res);
}

template <> hipblasStatus_t cublas_nrm2<double>(hipblasHandle_t hndl, int n, const double* x, int inc, double* res) {
    return hipblasDnrm2(hndl, n, x, inc, res);
}

template <>
hipblasStatus_t cublas_nrm2<float_complext>(hipblasHandle_t hndl, int n, const float_complext* x, int inc, float* res) {
    return hipblasScnrm2(hndl, n, (const hipComplex*)x, inc, res);
}

template <>
hipblasStatus_t cublas_nrm2<double_complext>(hipblasHandle_t hndl, int n, const double_complext* x, int inc,
                                            double* res) {
    return hipblasDznrm2(hndl, n, (const hipDoubleComplex*)x, inc, res);
}

// DOT
//

template <>
hipblasStatus_t cublas_dot<float>(hipblasHandle_t hndl, int n, const float* x, int incx, const float* y, int incy,
                                 float* res, bool cc) {
    return hipblasSdot(hndl, n, x, incx, y, incy, res);
}

template <>
hipblasStatus_t cublas_dot<double>(hipblasHandle_t hndl, int n, const double* x, int incx, const double* y, int incy,
                                  double* res, bool cc) {
    return hipblasDdot(hndl, n, x, incx, y, incy, res);
}

template <>
hipblasStatus_t cublas_dot<float_complext>(hipblasHandle_t hndl, int n, const float_complext* x, int incx,
                                          const float_complext* y, int incy, float_complext* res, bool cc) {
    if (cc)
        return hipblasCdotc(hndl, n, (const hipComplex*)x, incx, (const hipComplex*)y, incy, (hipComplex*)res);
    else
        return hipblasCdotu(hndl, n, (const hipComplex*)x, incx, (const hipComplex*)y, incy, (hipComplex*)res);
}

template <>
hipblasStatus_t cublas_dot<double_complext>(hipblasHandle_t hndl, int n, const double_complext* x, int incx,
                                           const double_complext* y, int incy, double_complext* res, bool cc) {
    if (cc)
        return hipblasZdotc(hndl, n, (const hipDoubleComplex*)x, incx, (const hipDoubleComplex*)y, incy,
                           (hipDoubleComplex*)res);
    else
        return hipblasZdotu(hndl, n, (const hipDoubleComplex*)x, incx, (const hipDoubleComplex*)y, incy,
                           (hipDoubleComplex*)res);
}

// AXPY
//

template <>
hipblasStatus_t cublas_axpy<float>(hipblasHandle_t hndl, int n, const float* a, const float* x, int incx, float* y,
                                  int incy) {
    return hipblasSaxpy(hndl, n, a, x, incx, y, incy);
}

template <>
hipblasStatus_t cublas_axpy<double>(hipblasHandle_t hndl, int n, const double* a, const double* x, int incx, double* y,
                                   int incy) {
    return hipblasDaxpy(hndl, n, a, x, incx, y, incy);
}

template <>
hipblasStatus_t cublas_axpy<float_complext>(hipblasHandle_t hndl, int n, const float_complext* a, const float_complext* x,
                                           int incx, float_complext* y, int incy) {
    return hipblasCaxpy(hndl, n, (const hipComplex*)a, (const hipComplex*)x, incx, (hipComplex*)y, incy);
}

template <>
hipblasStatus_t cublas_axpy<double_complext>(hipblasHandle_t hndl, int n, const double_complext* a,
                                            const double_complext* x, int incx, double_complext* y, int incy) {
    return hipblasZaxpy(hndl, n, (const hipDoubleComplex*)a, (const hipDoubleComplex*)x, incx, (hipDoubleComplex*)y, incy);
}

// SUM
//

template <> hipblasStatus_t cublas_asum<float>(hipblasHandle_t hndl, int n, const float* x, int incx, float* result) {
    return hipblasSasum(hndl, n, x, incx, result);
}

template <> hipblasStatus_t cublas_asum<double>(hipblasHandle_t hndl, int n, const double* x, int incx, double* result) {
    return hipblasDasum(hndl, n, x, incx, result);
}

template <>
hipblasStatus_t cublas_asum<float_complext>(hipblasHandle_t hndl, int n, const float_complext* x, int incx,
                                           float* result) {
    return hipblasScasum(hndl, n, (const hipComplex*)x, incx, result);
}

template <>
hipblasStatus_t cublas_asum<double_complext>(hipblasHandle_t hndl, int n, const double_complext* x, int incx,
                                            double* result) {
    return hipblasDzasum(hndl, n, (const hipDoubleComplex*)x, incx, result);
}

// AMIN
//

template <> hipblasStatus_t cublas_amin<float>(hipblasHandle_t hndl, int n, const float* x, int incx, int* result) {
    return hipblasIsamin(hndl, n, x, incx, result);
}

template <> hipblasStatus_t cublas_amin<double>(hipblasHandle_t hndl, int n, const double* x, int incx, int* result) {
    return hipblasIdamin(hndl, n, x, incx, result);
}

template <>
hipblasStatus_t cublas_amin<float_complext>(hipblasHandle_t hndl, int n, const float_complext* x, int incx, int* result) {
    return hipblasIcamin(hndl, n, (const hipComplex*)x, incx, result);
}

template <>
hipblasStatus_t cublas_amin<double_complext>(hipblasHandle_t hndl, int n, const double_complext* x, int incx,
                                            int* result) {
    return hipblasIzamin(hndl, n, (const hipDoubleComplex*)x, incx, result);
}

// AMAX
//

template <> hipblasStatus_t cublas_amax<float>(hipblasHandle_t hndl, int n, const float* x, int incx, int* result) {
    return hipblasIsamax(hndl, n, x, incx, result);
}

template <> hipblasStatus_t cublas_amax<double>(hipblasHandle_t hndl, int n, const double* x, int incx, int* result) {
    return hipblasIdamax(hndl, n, x, incx, result);
}

template <>
hipblasStatus_t cublas_amax<float_complext>(hipblasHandle_t hndl, int n, const float_complext* x, int incx, int* result) {
    return hipblasIcamax(hndl, n, (const hipComplex*)x, incx, result);
}

template <>
hipblasStatus_t cublas_amax<double_complext>(hipblasHandle_t hndl, int n, const double_complext* x, int incx,
                                            int* result) {
    return hipblasIzamax(hndl, n, (const hipDoubleComplex*)x, incx, result);
}

template <class T> typename realType<T>::Type nrm2(cuNDArray<T>* arr) {
    if (arr == 0x0)
        throw std::runtime_error("Gadgetron::nrm2(): Invalid input array");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    typedef typename realType<T>::Type REAL;
    REAL ret;

    // If number of elements in the array is greater than INT_MAX break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = arr->get_number_of_elements() / INT_MAX + 1;
    int remainder = arr->get_number_of_elements() - INT_MAX * (num_splits - 1);

    for (int ii = 0; ii < num_splits; ii++) {

        REAL val;

        CUBLAS_CALL(cublas_nrm2<T>(cudaDeviceManager::Instance()->lockHandle(device),
                                   (ii == num_splits - 1) ? remainder : INT_MAX, // n number of elements
                                   arr->get_data_ptr() + INT_MAX * ii, 1, &val));

        cudaDeviceManager::Instance()->unlockHandle(device);

        if (ii == 0)
            ret = val;
        else
            ret = sqrt(pow(ret, 2) + pow(val, 2));
    }
    return ret;
}

template <class T> T dot(cuNDArray<T>* arr1, cuNDArray<T>* arr2, bool cc) {
    if (arr1 == 0x0 || arr2 == 0x0)
        throw std::runtime_error("Gadgetron::dot(): Invalid input array");

    if (arr1->get_number_of_elements() != arr2->get_number_of_elements())
        throw std::runtime_error("Gadgetron::dot(): Array sizes mismatch");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    T ret;

    // If number of elements in the array is greater than INT_MAX break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = arr1->get_number_of_elements() / INT_MAX + 1;
    int remainder = arr1->get_number_of_elements() - INT_MAX * (num_splits - 1);

    for (int ii = 0; ii < num_splits; ii++) {

        T val;
        CUBLAS_CALL(cublas_dot(cudaDeviceManager::Instance()->lockHandle(device),
                               (ii == num_splits - 1) ? remainder : INT_MAX, // n number of elements
                               arr1->get_data_ptr() + INT_MAX * ii, 1, arr2->get_data_ptr() + INT_MAX * ii, 1, &val,
                               cc));

        cudaDeviceManager::Instance()->unlockHandle(device);
        if (ii == 0)
            ret = val;
        else
            ret += val;
    }

    return ret;
}

template <class T> void axpy(T a, cuNDArray<T>* x, cuNDArray<T>* y) {
    if (x == 0x0 || y == 0x0)
        throw std::runtime_error("Gadgetron::axpy(): Invalid input array");

    if (x->get_number_of_elements() != y->get_number_of_elements())
        throw std::runtime_error("Gadgetron::axpy(): Array sizes mismatch");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();

    // If number of elements in the array is greater than INT_MAX break it up and perform calculations this is done to
    // support large data arrays
    int num_splits = x->get_number_of_elements() / INT_MAX + 1;
    int remainder = x->get_number_of_elements() - INT_MAX * (num_splits - 1);

    for (int ii = 0; ii < num_splits; ii++) {

        CUBLAS_CALL(cublas_axpy(cudaDeviceManager::Instance()->lockHandle(device),
                                (ii == num_splits - 1) ? remainder : INT_MAX, &a, x->get_data_ptr() + INT_MAX * ii, 1,
                                y->get_data_ptr() + INT_MAX * ii, 1));

        cudaDeviceManager::Instance()->unlockHandle(device);
    }
}

template <class T> void axpy(T a, cuNDArray<complext<T>>* x, cuNDArray<complext<T>>* y) { axpy(complext<T>(a), x, y); }

template <class T> typename realType<T>::Type asum(cuNDArray<T>* x) {
    if (x == 0x0)
        throw std::runtime_error("Gadgetron::asum(): Invalid input array");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    typename realType<T>::Type result;

    // If number of elements in the array is greater than INT_MAX break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = x->get_number_of_elements() / INT_MAX + 1;
    int remainder = x->get_number_of_elements() - INT_MAX * (num_splits - 1);

    for (int ii = 0; ii < num_splits; ii++) {
        typename realType<T>::Type interim_result;

        CUBLAS_CALL(cublas_asum(cudaDeviceManager::Instance()->lockHandle(device),
                                (ii == num_splits - 1) ? remainder : INT_MAX, // n number of elements
                                x->get_data_ptr() + INT_MAX * ii, 1, &interim_result));

        cudaDeviceManager::Instance()->unlockHandle(device);

        if (ii == 0)
            result = interim_result;
        else
            result += interim_result;
    }
    return result;
}

template <typename T> size_t amin(cuNDArray<T>* x) {
    if (x == 0x0)
        throw std::runtime_error("Gadgetron::amin(): Invalid input array");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    size_t result;

    // If number of elements in the array is greater than INT_MAX break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = x->get_number_of_elements() / INT_MAX + 1;
    int remainder = x->get_number_of_elements() - INT_MAX * (num_splits - 1);

    for (int ii = 0; ii < num_splits; ii++) {

        int interim_result;

        CUBLAS_CALL(cublas_amin(cudaDeviceManager::Instance()->lockHandle(device),
                                (ii == num_splits - 1) ? remainder : INT_MAX, // n number of elements
                                x->get_data_ptr() + INT_MAX * ii, 1, &interim_result));

        cudaDeviceManager::Instance()->unlockHandle(device);

        if (ii == 0)
            result = (size_t)interim_result - 1;
        else if (((*x)[INT_MAX * ii + (size_t)interim_result - 1]) < ((*x)[result]))
            result = INT_MAX * ii + (size_t)interim_result - 1;

    }
   
    if (result > x->get_number_of_elements()) {
            throw std::runtime_error("Gadgetron::amax(): computed index is out of bounds");
        }
    return result; // result - 1;
}

template <typename T> size_t amax(cuNDArray<T>* x) {
    if (x == 0x0)
        throw std::runtime_error("Gadgetron::amax(): Invalid input array");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    size_t result;

    // If number of elements in the array is greater than INT_MAX break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = x->get_number_of_elements() / INT_MAX + 1;
    int remainder = x->get_number_of_elements() - INT_MAX * (num_splits - 1);

    for (int ii = 0; ii < num_splits; ii++) {

        int interim_result;
        CUBLAS_CALL(cublas_amax(
            cudaDeviceManager::Instance()->lockHandle(device),
            (ii == num_splits - 1) ? remainder : INT_MAX, // n number of elements (int)x->get_number_of_elements(),
            x->get_data_ptr() + INT_MAX * ii, 1, &interim_result));

        cudaDeviceManager::Instance()->unlockHandle(device);
       
        if (ii == 0)
            result = (size_t)interim_result - 1;
        else if ((((*x)[INT_MAX * ii + (size_t)interim_result - 1]) > ((*x)[result])))
            result = INT_MAX * ii + (size_t)interim_result - 1;

    }

    if (result > x->get_number_of_elements()) {
            throw std::runtime_error("Gadgetron::amax(): computed index is out of bounds");
        }
    return result; //(size_t)result - 1;
}
// Complex
template <typename T> size_t amin(cuNDArray<complext<T>>* x) {
    if (x == 0x0)
        throw std::runtime_error("Gadgetron::amin(): Invalid input array");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    size_t result;

    // If number of elements in the array is greater than INT_MAX break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = x->get_number_of_elements() / INT_MAX + 1;
    int remainder = x->get_number_of_elements() - INT_MAX * (num_splits - 1);
    complext<T> saved_value;

    for (int ii = 0; ii < num_splits; ii++) {

        int interim_result;

        CUBLAS_CALL(cublas_amin(cudaDeviceManager::Instance()->lockHandle(device),
                                (ii == num_splits - 1) ? remainder : INT_MAX, // n number of elements
                                x->get_data_ptr() + INT_MAX * ii, 1, &interim_result));

        cudaDeviceManager::Instance()->unlockHandle(device);

        auto interim_value = (*x)[INT_MAX * ii + (size_t)interim_result - 1];
        if (ii == 0)
            result = (size_t)interim_result - 1;
        else if ((abs(interim_value.real()) + abs(interim_value.imag())) <
                 (abs(saved_value.real()) + abs(saved_value.imag())))
            result = INT_MAX * ii + (size_t)interim_result - 1;

        saved_value = (*x)[result];

        
    }
    if (result > x->get_number_of_elements()) {
            throw std::runtime_error("Gadgetron::amin(): computed index is out of bounds");
        }
    
    return result; // result - 1;
}

template <typename T> size_t amax(cuNDArray<complext<T>>* x) {
    if (x == 0x0)
        throw std::runtime_error("Gadgetron::amax(): Invalid input array");

    int device = cudaDeviceManager::Instance()->getCurrentDevice();
    size_t result;

    // If number of elements in the array is greater than INT_MAX break it up and perform calculations this is done to
    // support large data arrays

    int num_splits = x->get_number_of_elements() / INT_MAX + 1;
    int remainder = x->get_number_of_elements() - INT_MAX * (num_splits - 1);
    complext<T> saved_value;
    for (int ii = 0; ii < num_splits; ii++) {

        int interim_result;
        CUBLAS_CALL(cublas_amax(
            cudaDeviceManager::Instance()->lockHandle(device),
            (ii == num_splits - 1) ? remainder : INT_MAX, // n number of elements (int)x->get_number_of_elements(),
            x->get_data_ptr() + INT_MAX * ii, 1, &interim_result));

        cudaDeviceManager::Instance()->unlockHandle(device);
        
        auto interim_value = (*x)[INT_MAX * ii + (size_t)interim_result - 1];
        
        if (ii == 0)
            result = (size_t)interim_result - 1;
        else if (abs(interim_value.real()) + abs(interim_value.imag()) >
                 abs(saved_value.real()) + abs(saved_value.imag()))
            result = INT_MAX * ii + (size_t)interim_result - 1;
        
        saved_value = (*x)[result];

        
    }
    if (result > x->get_number_of_elements()) {
            throw std::runtime_error("Gadgetron::amax(): computed index is out of bounds");
        }
    return result; //(size_t)result - 1;
}

std::string gadgetron_getCublasErrorString(hipblasStatus_t err) {
    switch (err) {
    case HIPBLAS_STATUS_NOT_INITIALIZED:
        return "NOT INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
        return "ALLOC FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
        return "INVALID VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
        return "ARCH MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
        return "MAPPING ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
        return "EXECUTION FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
        return "INTERNAL ERROR";
    case HIPBLAS_STATUS_SUCCESS:
        return "SUCCES";
    default:
        return "UNKNOWN CUBLAS ERROR";
    }
}

//
// Instantiation
//

template float dot(cuNDArray<float>*, cuNDArray<float>*, bool);
template float nrm2(cuNDArray<float>*);
template void axpy(float, cuNDArray<float>*, cuNDArray<float>*);
template size_t amin(cuNDArray<float>*);
template size_t amax(cuNDArray<float>*);
template float asum(cuNDArray<float>*);

template double dot(cuNDArray<double>*, cuNDArray<double>*, bool);
template double nrm2(cuNDArray<double>*);
template void axpy(double, cuNDArray<double>*, cuNDArray<double>*);
template size_t amin(cuNDArray<double>*);
template size_t amax(cuNDArray<double>*);
template double asum(cuNDArray<double>*);

template float_complext dot(cuNDArray<float_complext>*, cuNDArray<float_complext>*, bool);
template float nrm2(cuNDArray<float_complext>*);
template void axpy(float_complext, cuNDArray<float_complext>*, cuNDArray<float_complext>*);
template void axpy(float, cuNDArray<float_complext>*, cuNDArray<float_complext>*);
template size_t amin(cuNDArray<float_complext>*);
template size_t amax(cuNDArray<float_complext>*);
template float asum(cuNDArray<float_complext>*);

template double_complext dot(cuNDArray<double_complext>*, cuNDArray<double_complext>*, bool);
template double nrm2(cuNDArray<double_complext>*);
template void axpy(double_complext, cuNDArray<double_complext>*, cuNDArray<double_complext>*);
template void axpy(double, cuNDArray<double_complext>*, cuNDArray<double_complext>*);
template size_t amin(cuNDArray<double_complext>*);
template size_t amax(cuNDArray<double_complext>*);
template double asum(cuNDArray<double_complext>*);
} // namespace Gadgetron
